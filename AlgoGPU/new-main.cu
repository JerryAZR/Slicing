#include <iostream>
#include <string>
#include "triangle.cuh"
#include "slicer.cuh"
#include <vector>
#include <map>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include "golden.cuh"

#define BLOCK_SIZE 16

int main(int argc, char* argv[]) {

    std::string stl_file_name;
    std::vector<triangle> triangles;

    if (argc == 2) {
        stl_file_name = argv[1];
    }
    else if (argc > 2) {
        std::cout << "ERROR: Too many command line arguments" << std::endl;
    }

    read_stl(stl_file_name, triangles);
    int num_triangles = triangles.size();

    //load from host to device

    bool* outArray = (bool*)malloc(X_DIM * Y_DIM * NUM_LAYERS * sizeof(bool));
    bool* d_outArray;
    hipMalloc(&d_outArray, X_DIM * Y_DIM * NUM_LAYERS * sizeof(bool));

    triangle* d_triangles;
    hipMalloc(&d_triangles, num_triangles * sizeof(triangle));
    hipMemcpy(d_triangles, triangles.data(), num_triangles * sizeof(triangle), hipMemcpyHostToDevice);

    double* zmins;
    hipMalloc(&zmins, num_triangles * sizeof(double));

    int* h_index = (int*)malloc(num_triangles * sizeof(int));
    int* index;
    hipMalloc(&index, num_triangles * sizeof(int));


    hipError_t err = hipGetLastError();  // add
    if (err != hipSuccess) std::cout << "CUDA error 1: " << hipGetErrorString(err) << std::endl;

    int threadsPerBlock = THREADS_PER_BLOCK;
    int blocksPerGrid;

    blocksPerGrid = (num_triangles + threadsPerBlock - 1) / threadsPerBlock;
    triangle_sort << <blocksPerGrid, threadsPerBlock >> > (d_triangles, num_triangles, zmins, index);
    hipDeviceSynchronize();
    err = hipGetLastError();  // add
    if (err != hipSuccess) std::cout << "CUDA error 2: " << hipGetErrorString(err) << std::endl;

    thrust::device_ptr<double> d_zmins(zmins);
    thrust::device_ptr<int> d_index(index);

    thrust::sort_by_key(d_zmins, d_zmins + num_triangles, d_index);
    index = thrust::raw_pointer_cast(&d_index[0]);

    hipDeviceSynchronize();
    err = hipGetLastError();  // add
    if (err != hipSuccess) std::cout << "CUDA error 3: " << hipGetErrorString(err) << std::endl;


    blocksPerGrid = (Y_DIM * X_DIM + threadsPerBlock - 1) / threadsPerBlock;
    outputArray << <blocksPerGrid, threadsPerBlock >> > (d_triangles, num_triangles, d_outArray, index);

    hipDeviceSynchronize();
    err = hipGetLastError();  // add
    if (err != hipSuccess) std::cout << "CUDA error 4: " << hipGetErrorString(err) << std::endl;
    // Copy result from device memory to host memory
    hipMemcpy(outArray, d_outArray, X_DIM * Y_DIM * NUM_LAYERS * sizeof(bool), hipMemcpyDeviceToHost);

    err = hipGetLastError();  // add
    if (err != hipSuccess) std::cout << "CUDA error 5: " << hipGetErrorString(err) << std::endl;

    err = hipGetLastError();  // add
    if (err != hipSuccess) std::cout << "CUDA error 6: " << hipGetErrorString(err) << std::endl;
    /*
    std::cout << "begin verification" << std::endl;
    checkOutput(d_triangles, num_triangles, outArray);
    */
    hipFree(d_outArray);
    hipFree(d_triangles);
    hipFree(zmins);
    hipFree(index);

    return 0;
}