#include <iostream>
#include <string>
#include "triangle.cuh"
#include "slicer.cuh"
#include <vector>
#include <map>

#define BLOCK_SIZE 16

int main(int argc, char* argv[]) {
    std::string stl_file_name;
    std::vector<triangle> triangles;

    if (argc == 2) {
        stl_file_name = argv[1];
    }
    else if (argc > 2) {
        std::cout << "ERROR: Too many command line arguments" << std::endl;
    }

    read_stl(stl_file_name, triangles);
    int num_triangles = triangles.size();

    //load from host to device

    bool* outArray = (bool*)malloc(X_DIM * Y_DIM * NUM_LAYERS * sizeof(bool));
    bool* d_outArray;
    hipMalloc(&d_outArray, X_DIM * Y_DIM * NUM_LAYERS * sizeof(bool));

    triangle* d_triangles;
    hipMalloc(&d_triangles, num_triangles * sizeof(triangle));
    hipMemcpy(d_triangles, triangles.data(), num_triangles * sizeof(triangle), hipMemcpyHostToDevice);

    //bool* flagArray;
    //hipMalloc(&flagArray, X_DIM * Y_DIM * sizeof(bool));

    hipError_t err = hipGetLastError();  // add
    if (err != hipSuccess) std::cout << "CUDA error 1: " << hipGetErrorString(err) << std::endl;

    int threadsPerBlock = THREADS_PER_BLOCK;
    int blocksPerGrid;
    blocksPerGrid = (Y_DIM * X_DIM + threadsPerBlock - 1) / threadsPerBlock;

    outputArray << <blocksPerGrid, threadsPerBlock >> > (d_triangles, num_triangles, d_outArray);

    hipDeviceSynchronize();
    err = hipGetLastError();  // add
    if (err != hipSuccess) std::cout << "CUDA error 2: " << hipGetErrorString(err) << std::endl;
    // Copy result from device memory to host memory
    hipMemcpy(outArray, d_outArray, X_DIM * Y_DIM * NUM_LAYERS * sizeof(bool), hipMemcpyDeviceToHost);

    err = hipGetLastError();  // add
    if (err != hipSuccess) std::cout << "CUDA error 3: " << hipGetErrorString(err) << std::endl;

    hipFree(d_outArray);
    //hipFree(flagArray);
    hipFree(d_triangles);
    //hipFree(d_intersectArray);
    //hipFree(d_intersectArrayPre);

    err = hipGetLastError();  // add
    if (err != hipSuccess) std::cout << "CUDA error 4: " << hipGetErrorString(err) << std::endl;
/*
    for (int y = Y_DIM; y > 0; y--) {
        for (int x = 0; x < X_DIM; x++) {
            if (outArray[0 * Y_DIM * X_DIM + y * X_DIM + x]) std::cout << "x";
            else std::cout << " ";
        }
        std::cout << std::endl;
    }
*/
    return 0;
}
