#include "hip/hip_runtime.h"
#include "slicer.cuh"
#include "triangle.cuh"
#include <thrust/functional.h>

__device__ __forceinline__ void triangleCopy(void* src, void* dest, int id);
__device__ __forceinline__ double min3(double a, double b, double c);
__device__ __forceinline__ double max3(double a, double b, double c);
__device__ __forceinline__ char atomicAdd(char* address, char val);

__global__ void rectTriIntersection(triangle* tri_global, size_t num_tri, bool* out) {
    size_t idx = (size_t)blockDim.x * (size_t)blockIdx.x + (size_t)threadIdx.x;
    size_t num_per_thread = num_tri / (NUM_BLOCKS << LOG_THREADS) + 1;
    size_t base_idx = num_per_thread * idx;

    // iterate over all triangles assigned to this thread.
    for (size_t i = 0; i < num_per_thread; i++) {
        // Compute bounding box
        if (base_idx >= num_tri) break;
        triangle t = tri_global[base_idx++]; // Using post-increment;
        int xMin = (int)(min3(t.p1.x, t.p2.x, t.p3.x) / RESOLUTION);
        int yMin = (int)(min3(t.p1.y, t.p2.y, t.p3.y) / RESOLUTION);
        int xMax = __double2int_ru(max3(t.p1.x, t.p2.x, t.p3.x) / RESOLUTION);
        int yMax = __double2int_ru(max3(t.p1.y, t.p2.y, t.p3.y) / RESOLUTION);
        // Make sure the bounds are inside the supported space
        xMax = min(xMax, X_MAX);
        xMin = max(xMin, X_MIN);
        yMax = min(yMax, Y_MAX);
        yMin = max(yMin, Y_MIN);
        // iterate over all pixels inside the bounding box
        // Will likely cause (lots of) wrap divergence, but we'll deal with that later
        int x = xMin;
        int y = yMin;
        while (y <= yMax) {
            layer_t curr_intersection = pixelRayIntersection(t, x, y);
            if (curr_intersection >= 0 && curr_intersection < NUM_LAYERS) {
                // Found a valid intersection
                int x_idx = x + (X_DIM >> 1);
                int y_idx = y + (Y_DIM >> 1);
                char* temp_ptr = (char*) (out + curr_intersection*X_DIM*Y_DIM + y_idx*X_DIM + x_idx);
                atomicAdd(temp_ptr, 1);
            }
            // update coords
            bool nextLine = (x == xMax);
            y += (int)nextLine;
            x = nextLine ? xMin : (x+1);
        }
    }
}

__global__
void layerExtraction(bool* out, layer_t start) {
    size_t idx = (size_t)blockDim.x * (size_t)blockIdx.x + (size_t)threadIdx.x;
    bool isInside = false;
    char* out_ptr = (char*) (out + idx);
    char intersection_count;
    for (size_t i = start; i < NUM_LAYERS; i++) {
        intersection_count = out_ptr[i*X_DIM*Y_DIM];
        bool flip = (bool)(intersection_count & 1);
        bool intersect = (intersection_count > 0);
        out_ptr[i*X_DIM*Y_DIM] = (char) (isInside || intersect);
        isInside = isInside ^ flip;
    }
}

/**
 * pixelRayIntersection: helper function, computes the intersection of given triangle and pixel ray
 * Inputs:
 *      t -- input triangle
 *      x, y -- coordinates of the input pixel ray
 * Returns:
 *      The layer on which they intersect, or -1 if no intersection
 */
__device__ __forceinline__
layer_t pixelRayIntersection(triangle t, int x, int y) {
    /*
    Let A, B, C be the 3 vertices of the given triangle
    Let S(x,y,z) be the intersection, where x,y are given
    We want to find some a, b such that AS = a*AB + b*AC
    If a >= 0, b >= 0, and a+b <= 1, S is a valid intersection.
    */

    double x_pos = x * RESOLUTION;
    double y_pos = y * RESOLUTION;

    // double x_max = max3(t.p1.x, t.p2.x, t.p3.x);
    // double x_min = min3(t.p1.x, t.p2.x, t.p3.x);

    // if (x_pos < x_min || x_pos > x_max) return NONE;

    double x_d = x_pos - t.p1.x;
    double y_d = y_pos - t.p1.y;

    double x1 = t.p2.x - t.p1.x;
    double y1 = t.p2.y - t.p1.y;
    double z1 = t.p2.z - t.p1.z;

    double x2 = t.p3.x - t.p1.x;
    double y2 = t.p3.y - t.p1.y;
    double z2 = t.p3.z - t.p1.z;
    double a = (x_d * y2 - x2 * y_d) / (x1 * y2 - x2 * y1);
    double b = (x_d * y1 - x1 * y_d) / (x2 * y1 - x1 * y2);
    bool inside = (a >= 0) && (b >= 0) && (a+b <= 1);
    double intersection = (a * z1 + b * z2) + t.p1.z;
    // // divide by layer width
    layer_t layer = inside ? (intersection / RESOLUTION) : (layer_t)(-1);
    return layer;
}
 
// Copy (THREADS_PER_BLOCK) triangles from src to dest
// Achieves 100% memory efficiency
__device__ __forceinline__
void triangleCopy(void* src, void* dest, int id) {
    copy_unit_t* src_ptr = (copy_unit_t*) src;
    copy_unit_t* dest_ptr = (copy_unit_t*) dest;

    #pragma unroll
    for (int d = 0; d < unit_per_tri; d++) {
        size_t offset = d * THREADS_PER_BLOCK;
        dest_ptr[id + offset] = src_ptr[id + offset];
    }
}

__device__ __forceinline__
double min3(double a, double b, double c) {
    // thrust::minimum<double> min;
    return min(a, min(b, c));
}

__device__ __forceinline__
double max3(double a, double b, double c) {
    // thrust::maximum<double> max;
    return max(a, max(b, c));
}

__device__ __forceinline__
char atomicAdd(char* address, char val) {
    // *address = *address + val;
    // return 0;
    size_t addr_offset = (size_t) address & 3;
    auto* base_address = (unsigned int*) ((size_t) address - addr_offset);
    unsigned int long_val = (unsigned int) val << (8 * addr_offset);
    unsigned int long_old = atomicAdd(base_address, long_val);

    // Overflow check. skipped for simplicity.
    // if (addr_offset == 3) {
    //     return (char) (long_old >> 24);
    // } else {
    //     // bits that represent the char value within long_val
    //     unsigned int mask = 0x000000ff << (8 * addr_offset);
    //     unsigned int masked_old = long_old & mask;
    //     // isolate the bits that represent the char value within long_old, add the long_val to that,
    //     // then re-isolate by excluding bits that represent the char value
    //     unsigned int overflow = (masked_old + long_val) & ~mask;
    //     if (overflow) {
    //         atomicSub(base_address, overflow);
    //     }
    //     return (char) (masked_old >> 8 * addr_offset);
    // }

    return (char) ((long_old >> 8 * addr_offset) & 0xff);
}
