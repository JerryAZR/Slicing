#include <iostream>
#include <string>
#include "triangle.cuh"
#include "slicer.cuh"
#include <vector>

int main(int argc, char* argv[]) {
    std::string stl_file_name;
    std::vector<triangle> triangles;

    if (argc == 2) {
        stl_file_name = argv[1];
    } else if (argc > 2) {
        std::cout << "ERROR: Too many command line arguments" << std::endl;
    }

    read_stl(stl_file_name,triangles);
    int num_triangles = triangles.size();
    triangle* triangles_dev;
    // all[z][y][x]
    bool* all = (bool*)malloc(NUM_LAYERS * Y_DIM * X_DIM * sizeof(bool));
    bool* all_dev;
    size_t size = NUM_LAYERS * Y_DIM * X_DIM * sizeof(bool);
    hipMalloc(&all_dev, size);
    hipMalloc(&triangles_dev, num_triangles * sizeof(triangle));
    //hipMemcpy(all_dev, &all[0][0][0], size, hipMemcpyHostToDevice); // unnecessary
    hipMemcpy(triangles_dev, triangles.data(), num_triangles * sizeof(triangle), hipMemcpyHostToDevice);

    hipError_t err = hipGetLastError();  // add
    if (err != hipSuccess) std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl;

    int threadsPerBlock = THREADS_PER_BLOCK;
    int blocksPerGrid;

    blocksPerGrid = (Y_DIM * X_DIM + threadsPerBlock - 1) / threadsPerBlock;
    pps<<<blocksPerGrid, threadsPerBlock>>>(&triangles_dev[0], num_triangles, all_dev);
    hipDeviceSynchronize();
    err = hipGetLastError();  // add
    if (err != hipSuccess) std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl;

    // Copy result from device memory to host memory
    hipMemcpy(all, all_dev, size, hipMemcpyDeviceToHost);
    err = hipGetLastError();  // add
    if (err != hipSuccess) std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl;

    free(all);
    hipFree(all_dev);
    hipFree(triangles_dev);

    return 0;
}
