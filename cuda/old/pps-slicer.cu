#include "hip/hip_runtime.h"
#include "slicer.cuh"
#include <thrust/sort.h>
#include <thrust/functional.h>
#include <stdio.h>

__global__
void pps(triangle* triangles_global, size_t num_triangles, bool* out) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    // printf("starting thread %d\n", idx);
    int y_idx = idx / X_DIM;
    // if (y >= Y_DIM) return;
    int x_idx = idx % X_DIM;
    int x = x_idx - (X_DIM / 2);
    int y = y_idx - (Y_DIM / 2);

    // Copy triangles to shared memory
    // Each block has a shared memory storing some triangles.
    __shared__ triangle tri_base[THREADS_PER_BLOCK];
    triangle* triangles = (triangle*) tri_base;
    size_t num_iters = num_triangles / THREADS_PER_BLOCK;
    int length = 0;
    __shared__ layer_t layers_shared[THREADS_PER_BLOCK][MAX_TRUNK_SIZE+1];
    layer_t* layers = &layers_shared[threadIdx.x][0];
    for (size_t i = 0; i < num_iters; i++) {
        triangles[threadIdx.x] = triangles_global[threadIdx.x + (i * THREADS_PER_BLOCK)];
        // Wait for other threads to complete;
        __syncthreads();
        if (y_idx < Y_DIM) {
            for (size_t tri_idx = 0; tri_idx < THREADS_PER_BLOCK; tri_idx++) {
                layer_t intersection = pixelRayIntersection(triangles[tri_idx], x, y);
                if (intersection != NONE) {
                    layers[length] = intersection;
                    length++;
                }
            }
        }
        __syncthreads();
    }
    size_t remaining = num_triangles - (num_iters * THREADS_PER_BLOCK);
    if (threadIdx.x < remaining) {
        triangles[threadIdx.x] = triangles_global[threadIdx.x + (num_iters * THREADS_PER_BLOCK)];
    }
    __syncthreads();
    if (remaining && y_idx < Y_DIM) {
        for (size_t tri_idx = 0; tri_idx < remaining; tri_idx++) {
            layer_t intersection = pixelRayIntersection(triangles[tri_idx], x, y);
            if (intersection != NONE) {
                layers[length] = intersection;
                length++;
            }
        }
    }

    if (y_idx >= Y_DIM) return;

    thrust::sort(thrust::device, &layers[0], &layers[length]);
    layers[length] = NUM_LAYERS;
    if (length > MAX_TRUNK_SIZE) 
        printf("Error: Too many intersections.\n \
                Please increase MAX_TRUNK_SIZE in slicer.cuh and recompile.\n");

    bool flag = false;
    int layerIdx = 0;
    for (layer_t z = 0; z < NUM_LAYERS; z++) {
        // If intersect
        while (layers[layerIdx] < z) layerIdx++;
        bool intersect = (z == layers[layerIdx]);
        flag = (bool) (layerIdx & 1);
        out[z*Y_DIM*X_DIM + y_idx*X_DIM + x_idx] = intersect || flag;
    }
}

/**
 * pixelRayIntersection: helper function, computes the intersection of given triangle and pixel ray
 * Inputs:
 *      t -- input triangle
 *      x, y -- coordinates of the input pixel ray
 * Returns:
 *      The layer on which they intersect, or -1 if no intersection
 */
__device__ __forceinline__
layer_t pixelRayIntersection(triangle t, int x, int y) {
    /*
    Let A, B, C be the 3 vertices of the given triangle
    Let S(x,y,z) be the intersection, where x,y are given
    We want to find some a, b such that AS = a*AB + b*AC
    If a >= 0, b >= 0, and a+b <= 1, S is a valid intersection.
    */

    double x_max = max(t.p1.x, max(t.p2.x, t.p3.x));
    double x_min = min(t.p1.x, min(t.p2.x, t.p3.x));
    double y_max = max(t.p1.y, max(t.p2.y, t.p3.y));
    double y_min = min(t.p1.y, min(t.p2.y, t.p3.y));

    double x_pos = x * RESOLUTION;
    double y_pos = y * RESOLUTION;
    if ((x_pos < x_min) || (x_pos > x_max) || (y_pos < y_min) || (y_pos > y_max)) return NONE;

    double x_d = x_pos - t.p1.x;
    double y_d = y_pos - t.p1.y;

    double x1 = t.p2.x - t.p1.x;
    double y1 = t.p2.y - t.p1.y;
    double z1 = t.p2.z - t.p1.z;

    double x2 = t.p3.x - t.p1.x;
    double y2 = t.p3.y - t.p1.y;
    double z2 = t.p3.z - t.p1.z;
    double a = (x_d * y2 - x2 * y_d) / (x1 * y2 - x2 * y1);
    double b = (x_d * y1 - x1 * y_d) / (x2 * y1 - x1 * y2);
    bool inside = (a >= 0) && (b >= 0) && (a+b <= 1);
    double intersection = (a * z1 + b * z2) + t.p1.z;
    // // divide by layer width
    layer_t layer = inside ? (intersection / RESOLUTION) : NONE;
    return layer;
}
