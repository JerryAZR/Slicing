#include <thrust/sort.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <algorithm>
#include <iostream>
#include <string>
#include <vector>
#include <utility>
#include "triangle.cuh"
#include "slicer.cuh"
#include "golden.cuh"

using std::pair;

int main(int argc, char* argv[]) {
    std::string stl_file_name;
    std::vector<triangle> small_tri;

    if (argc == 2) {
        stl_file_name = argv[1];
    } else if (argc > 2) {
        std::cout << "ERROR: Too many command line arguments" << std::endl;
        return 0;
    } else {
        std::cout << "ERROR: Too few command line arguments" << std::endl;
        return 0;
    }

    read_stl(stl_file_name, small_tri);
    size_t num_small = small_tri.size();

    triangle* small_tri_dev;
    hipMalloc(&small_tri_dev, num_small * sizeof(triangle));
    hipMemcpy(small_tri_dev, small_tri.data(), num_small * sizeof(triangle), hipMemcpyHostToDevice);

    layer_t* intersections_large;
    hipMalloc(&intersections_large, Y_DIM * X_DIM * NUM_LAYERS * sizeof(layer_t));

    size_t* trunk_length;
    hipMalloc(&trunk_length, Y_DIM * X_DIM * sizeof(size_t));

    // out[y][x][z]
    bool* out = (bool*)malloc(NUM_LAYERS * Y_DIM * X_DIM * sizeof(bool));
    bool* out_dev;
    hipMalloc(&out_dev, Y_DIM * X_DIM * NUM_LAYERS * sizeof(bool));
    hipMemset(out_dev, 0, Y_DIM * X_DIM * NUM_LAYERS * sizeof(bool));

    double* z_mins_dev;
    hipMalloc(&z_mins_dev, num_small * sizeof(double));

    int threadsPerBlock = THREADS_PER_BLOCK;
    int numBlocks;

    numBlocks = (Y_DIM * X_DIM + threadsPerBlock - 1) / threadsPerBlock;

    GPUsort(small_tri_dev, num_small, z_mins_dev);

    hipDeviceSynchronize();

    smallTriIntersection<<<numBlocks, threadsPerBlock>>>(small_tri_dev, z_mins_dev, num_small, out_dev);

    hipDeviceSynchronize();

    hipMemcpy(out, out_dev, Y_DIM * X_DIM * NUM_LAYERS * sizeof(bool), hipMemcpyDeviceToHost);

    free(out);
    hipFree(small_tri_dev);
    hipFree(z_mins_dev);
    hipFree(out_dev);

    return 0;
}