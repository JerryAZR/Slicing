#include <thrust/sort.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <algorithm>
#include <iostream>
#include <string>
#include <vector>
#include <utility>
#include "triangle.cuh"
#include "slicer.cuh"
#include "golden.cuh"

using std::pair;

int main(int argc, char* argv[]) {
    std::string stl_file_name;
    std::vector<triangle> small_tri;
    std::vector<triangle> large_tri;
    std::vector<double> z_mins_vect;

    if (argc == 2) {
        stl_file_name = argv[1];
    } else if (argc > 2) {
        std::cout << "ERROR: Too many command line arguments" << std::endl;
        return 0;
    } else {
        std::cout << "ERROR: Too few command line arguments" << std::endl;
        return 0;
    }

    preprocess_stl(stl_file_name, small_tri, large_tri, z_mins_vect);
    size_t num_small = small_tri.size();
    size_t num_large = large_tri.size();

    triangle* small_tri_dev;
    hipMalloc(&small_tri_dev, num_small * sizeof(triangle));
    hipMemcpy(small_tri_dev, small_tri.data(), num_small * sizeof(triangle), hipMemcpyHostToDevice);

    triangle* large_tri_dev;
    hipMalloc(&large_tri_dev, num_large * sizeof(triangle));
    hipMemcpy(large_tri_dev, large_tri.data(), num_large * sizeof(triangle), hipMemcpyHostToDevice);

    layer_t* intersections_large;
    hipMalloc(&intersections_large, Y_DIM * X_DIM * NUM_LAYERS * sizeof(layer_t));

    size_t* trunk_length;
    hipMalloc(&trunk_length, Y_DIM * X_DIM * sizeof(size_t));

    // out[y][x][z]
    bool* out = (bool*)malloc(NUM_LAYERS * Y_DIM * X_DIM * sizeof(bool));
    bool* out_dev;
    hipMalloc(&out_dev, Y_DIM * X_DIM * NUM_LAYERS * sizeof(bool));
    hipMemset(out_dev, 0, Y_DIM * X_DIM * NUM_LAYERS * sizeof(bool));

    double* z_mins_dev;
    hipMalloc(&z_mins_dev, num_small * sizeof(double));
    hipMemcpy(z_mins_dev, z_mins_vect.data(), num_small * sizeof(double), hipMemcpyHostToDevice);

    int threadsPerBlock = THREADS_PER_BLOCK;
    int numBlocks;

    numBlocks = (Y_DIM * X_DIM + threadsPerBlock - 1) / threadsPerBlock;

    largeTriIntersection<<<numBlocks, threadsPerBlock>>>(large_tri_dev, num_large, intersections_large, trunk_length);
    thrust::sort_by_key(thrust::device, z_mins_dev, z_mins_dev + num_small, small_tri_dev);

    hipDeviceSynchronize();

    smallTriIntersection<<<numBlocks, threadsPerBlock>>>(small_tri_dev, z_mins_dev, num_small, intersections_large, trunk_length, out_dev);

    hipDeviceSynchronize();

    hipMemcpy(out, out_dev, Y_DIM * X_DIM * NUM_LAYERS * sizeof(bool), hipMemcpyDeviceToHost);

    free(out);
    hipFree(large_tri_dev);
    hipFree(small_tri_dev);
    hipFree(intersections_large);
    hipFree(trunk_length);

    return 0;
}