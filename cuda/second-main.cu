#include <thrust/sort.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <algorithm>
#include <iostream>
#include <string>
#include <vector>
#include <utility>
#include "triangle.cuh"
#include "slicer.cuh"
#include "golden.cuh"

using std::pair;

int main(int argc, char* argv[]) {
    std::string stl_file_name;
    std::vector<triangle> small_tri;
    std::vector<triangle> large_tri;
    std::vector<double> z_mins_vect;

    if (argc == 2) {
        stl_file_name = argv[1];
    } else if (argc > 2) {
        std::cout << "ERROR: Too many command line arguments" << std::endl;
        return 0;
    } else {
        std::cout << "ERROR: Too few command line arguments" << std::endl;
        return 0;
    }

    preprocess_stl(stl_file_name, small_tri, large_tri, z_mins_vect);
    size_t num_small = small_tri.size();
    size_t num_large = large_tri.size();

    triangle* small_tri_dev;
    hipMalloc(&small_tri_dev, num_small * sizeof(triangle));
    hipMemcpy(small_tri_dev, small_tri.data(), num_small * sizeof(triangle), hipMemcpyHostToDevice);

    triangle* large_tri_dev;
    hipMalloc(&large_tri_dev, num_large * sizeof(triangle));
    hipMemcpy(large_tri_dev, large_tri.data(), num_large * sizeof(triangle), hipMemcpyHostToDevice);

    layer_t* intersections_large;
    hipMalloc(&intersections_large, Y_DIM * X_DIM * NUM_LAYERS * sizeof(layer_t));

    size_t* trunk_length;
    hipMalloc(&trunk_length, Y_DIM * X_DIM * sizeof(size_t));

    // out[y][x][z]
    bool* out = (bool*)malloc(NUM_LAYERS * Y_DIM * X_DIM * sizeof(bool));
    bool* out_dev;
    hipMalloc(&out_dev, Y_DIM * X_DIM * NUM_LAYERS * sizeof(bool));
    hipMemset(out_dev, 0, Y_DIM * X_DIM * NUM_LAYERS * sizeof(bool));

    double* z_mins_dev;
    hipMalloc(&z_mins_dev, num_small * sizeof(double));
    hipMemcpy(z_mins_dev, z_mins_vect.data(), num_small * sizeof(double), hipMemcpyHostToDevice);

    int threadsPerBlock = THREADS_PER_BLOCK;
    int numBlocks;

    numBlocks = (Y_DIM * X_DIM + threadsPerBlock - 1) / threadsPerBlock;

    largeTriIntersection<<<numBlocks, threadsPerBlock>>>(large_tri_dev, num_large, intersections_large, trunk_length);
    thrust::sort_by_key(thrust::device, z_mins_dev, z_mins_dev + num_small, small_tri_dev);

    hipDeviceSynchronize();

    smallTriIntersection<<<numBlocks, threadsPerBlock>>>(small_tri_dev, z_mins_dev, num_small, intersections_large, trunk_length, out_dev);

    hipDeviceSynchronize();

    hipMemcpy(out, out_dev, Y_DIM * X_DIM * NUM_LAYERS * sizeof(bool), hipMemcpyDeviceToHost);

    bool* out_reshaped = (bool*)malloc(NUM_LAYERS * Y_DIM * X_DIM * sizeof(bool));

    for (int z = 0; z < NUM_LAYERS; z++) {
        for (int y = Y_DIM-1; y >= 0; y--) {
            for (int x = 0; x < X_DIM; x++) {
                // if (out[y][x][z]) std::cout << "XX";
                // else std::cout << "  ";

                // out_reshaped[z][y][x] = out[y][x][z];
                out_reshaped[z * X_DIM * Y_DIM + y * X_DIM + x] =
                    out[y * X_DIM * NUM_LAYERS + x * NUM_LAYERS + z]; 
            }
            // std::cout << std::endl;
        }
        // std::cout << std::endl << std::endl;
    }

    free(out);
    free(out_reshaped);
    hipFree(large_tri_dev);
    hipFree(small_tri_dev);
    hipFree(intersections_large);
    hipFree(trunk_length);

    return 0;
}