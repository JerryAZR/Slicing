#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
// #include <format>
#include "triangle.cuh"
#include "slicer.cuh"
#include "golden.cuh"
#include "bitmap.cuh"
#include <vector>
#include <chrono>
#define NOW (std::chrono::high_resolution_clock::now())

typedef std::chrono::time_point<std::chrono::high_resolution_clock> chrono_t;

void timer_checkpoint(chrono_t & checkpoint) {
#ifdef TEST
    chrono_t end = NOW;
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - checkpoint);
    std::cout << (double)duration.count()/1000 << "ms" << std::endl;
    checkpoint = end;
#else
    std::cout << std::endl;
#endif
}

double get_duration_ms(chrono_t checkpoint) {
    chrono_t end = NOW;
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - checkpoint);
    return ((double)duration.count())/1000;
}

void print_ms(double t) {
    unsigned long t_int = (unsigned long)t;
    unsigned ms = t_int % 1000; t_int = t_int / 1000;
    unsigned s = t_int % 60; t_int = t_int / 60;
    unsigned min = t_int % 60;
    unsigned hour = t_int / 60;
    printf("%u:%02u:%02u.%03u", hour, min, s, ms);
}

void checkCudaError() {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl;
        exit(1);
    }
}

int main(int argc, char* argv[]) {
    std::string stl_file_name;
    std::vector<triangle> triangles;
    std::vector<std::vector<double>> point_array(9);
    std::cout << "Block height is " << BBOX_BLOCK_HEIGHT << std::endl;

    if (argc == 2) {
        stl_file_name = argv[1];
    } else if (argc > 2) {
        std::cout << "ERROR: Too many command line arguments" << std::endl;
    }

    chrono_t start = NOW;

    load_point_array(stl_file_name, point_array, triangles);

    std::cout << "Reading STL file...                   ";
    timer_checkpoint(start);
    std::cout << "Allocating device memory...           ";

    int num_triangles = triangles.size();
    triangle* triangles_dev;
    double* points_dev;
    // all[z][y][x]
#if (COMPRESSION_ONLY == 0)
    double decompression_time = 0.0;
#ifdef TEST
    bool* all = (bool*)malloc(NUM_LAYERS * Y_DIM * X_DIM * sizeof(bool));
#else
    bool* all = (bool*)malloc(BBOX_BLOCK_HEIGHT * Y_DIM * X_DIM * sizeof(bool));
#endif
#endif
    unsigned* trunks_dev;
    hipMalloc(&trunks_dev, BBOX_BLOCK_HEIGHT * Y_DIM * MAX_TRUNK_SIZE * sizeof(unsigned));
    unsigned* trunks_out;
    hipMalloc(&trunks_out, BBOX_BLOCK_HEIGHT * Y_DIM * MAX_TRUNK_SIZE * sizeof(unsigned));
    unsigned* trunk_length;
    hipMalloc(&trunk_length, BBOX_BLOCK_HEIGHT * Y_DIM * sizeof(unsigned));
    hipMemset(trunk_length, 0, BBOX_BLOCK_HEIGHT * Y_DIM * sizeof(unsigned));

#ifdef TEST
    unsigned* trunks_host = (unsigned*)malloc(NUM_LAYERS * MAX_TRUNK_SIZE * Y_DIM * sizeof(unsigned));
#else
    unsigned* trunks_host = (unsigned*)malloc(BBOX_BLOCK_HEIGHT * MAX_TRUNK_SIZE * Y_DIM * sizeof(unsigned));
#endif
    hipMalloc(&triangles_dev, num_triangles * sizeof(triangle));
    hipMemcpy(triangles_dev, triangles.data(), num_triangles * sizeof(triangle), hipMemcpyHostToDevice);
    
    hipMalloc(&points_dev, num_triangles * sizeof(triangle));
    size_t temp_offset = 0;
    for (int i = 0; i < 9; i++) {
        hipMemcpy(points_dev + temp_offset, point_array[i].data(),
                    num_triangles * sizeof(double), hipMemcpyHostToDevice);
        temp_offset += num_triangles;
    }
    hipError_t err = hipGetLastError();  // add
    if (err != hipSuccess) {
        std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl;
        return 1;
    }

    timer_checkpoint(start);
    std::cout << "Slicing...                            " << std::endl;
    size_t compressed_model_size = 0;
    for (unsigned layer_idx = 0; layer_idx < NUM_LAYERS; layer_idx += BBOX_BLOCK_HEIGHT) {
        rectTriIntersection<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>
            (points_dev, num_triangles, trunks_dev, trunk_length, layer_idx);
        hipDeviceSynchronize();
        checkCudaError();
        unsigned max_length = thrust::reduce(thrust::device,
            trunk_length, trunk_length+Y_DIM*BBOX_BLOCK_HEIGHT, 0, thrust::maximum<unsigned>());
        max_length += 2; // Max number of runs + zero terminate
        if (max_length > MAX_TRUNK_SIZE) {
            std::cout << "too many intersections" << std::endl; return 0;
        }
        size_t blocksPerGrid = (Y_DIM * BBOX_BLOCK_HEIGHT + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        trunk_compress<<<blocksPerGrid, THREADS_PER_BLOCK>>>(trunks_dev, trunk_length, trunks_out, max_length);
        hipDeviceSynchronize();
        checkCudaError();
        size_t copy_layers = (layer_idx + BBOX_BLOCK_HEIGHT) < NUM_LAYERS ? BBOX_BLOCK_HEIGHT : NUM_LAYERS - layer_idx;
        size_t copy_size = copy_layers * Y_DIM * max_length * sizeof(unsigned);
        compressed_model_size += copy_size;
        unsigned* trunks_addr = &trunks_host[0];
        hipMemcpy(trunks_addr, trunks_out, copy_size, hipMemcpyDeviceToHost);
        hipMemset(trunk_length, 0, BBOX_BLOCK_HEIGHT * Y_DIM * sizeof(unsigned));
        hipDeviceSynchronize();
        checkCudaError();
    #if (COMPRESSION_ONLY == 0)
    #ifdef TEST
        bool* out_addr = &all[layer_idx*X_DIM*Y_DIM];
    #else
        bool* out_addr = &all[0];
    #endif
        decompression_time += rleDecode(trunks_addr, out_addr, copy_layers, max_length);
    #endif
        double elapsed_time = get_duration_ms(start);
        double estimate = elapsed_time / layer_idx * NUM_LAYERS;
        printf("Progress: %2.2f%%. Time: ", ((double)layer_idx*100)/NUM_LAYERS);
        print_ms(elapsed_time);
        printf(" / ");
        print_ms(estimate);
        printf("\n");
    }
    std::cout << std::endl;

    timer_checkpoint(start);
    std::cout << "Compressed model size: " << compressed_model_size << " bytes." << std::endl;
    hipFree(trunk_length);
    hipFree(points_dev);
    hipFree(trunks_dev);
    hipFree(trunks_out);

#if (COMPRESSION_ONLY == 0)
    std::cout << "Total decompression time: " << decompression_time << "ms" << std::endl;
#ifdef TEST
    // checkOutput(triangles_dev, num_triangles, all);
#if (WRITE_BMP == 1)
    Pixel black = BLACK;
    Pixel white = WHITE;
    const char outDir[] = "bmp";
    char fname[128];
    for (int z = 0; z < NUM_LAYERS; z++) {
        sprintf(fname, "%s/layer_%d.bmp", outDir, z);
        std::ofstream outfile(fname, std::ios::out | std::ios::binary);
        // Write BMP header
        BmpHeader header;
        header.setDim(X_DIM, Y_DIM);
        header.setRes(RESOLUTION);
        outfile.write((char*)&header, HEADER_SIZE);
        
        for (int y = 0; y < Y_DIM; y++) {
            for (int x = 0; x < X_DIM; x++) {
                if (all[z*X_DIM*Y_DIM + y*X_DIM + x])
                    outfile.write((char*) &black, 3);
                else
                    outfile.write((char*) &white, 3);
            }
        }
        std::cout << "Writing to output file...  "<< z+1 << "/" << NUM_LAYERS << "\r";
        outfile.close();
    }
    std::cout << std::endl;
#endif
    free(all);
#endif
#endif

    hipFree(triangles_dev);
    free(trunks_host);

    return 0;
}
