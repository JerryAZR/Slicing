#include <iostream>
#include <fstream>
#include <string>
#include "triangle.cuh"
#include "slicer.cuh"
#include "golden.cuh"
#include <vector>
#include <chrono>
#define NOW (std::chrono::high_resolution_clock::now())

typedef std::chrono::time_point<std::chrono::high_resolution_clock> chrono_t;

void timer_checkpoint(chrono_t & checkpoint) {
#ifdef TEST
    chrono_t end = NOW;
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - checkpoint);
    std::cout << (double)duration.count()/1000 << "ms" << std::endl;
    checkpoint = end;
#else
    std::cout << std::endl;
#endif
}

void checkCudaError() {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl;
        exit(1);
    }
}

int main(int argc, char* argv[]) {
    std::string stl_file_name;
    std::vector<triangle> triangles;
    std::vector<std::vector<double>> point_array(9);
    std::cout << "Block height is " << BBOX_BLOCK_HEIGHT << std::endl;

    if (argc == 2) {
        stl_file_name = argv[1];
    } else if (argc > 2) {
        std::cout << "ERROR: Too many command line arguments" << std::endl;
    }

    chrono_t start = NOW;

    load_point_array(stl_file_name, point_array, triangles);

    std::cout << "Reading STL file...                   ";
    timer_checkpoint(start);
    std::cout << "Allocating device memory...           ";

    int num_triangles = triangles.size();
    triangle* triangles_dev;
    double* points_dev;
    // all[z][y][x]
#if (COMPRESSION_ONLY == 0)
    double decompression_time = 0.0;
#ifdef TEST
    bool* all = (bool*)malloc(NUM_LAYERS * Y_DIM * X_DIM * sizeof(bool));
#else
    bool* all = (bool*)malloc(BBOX_BLOCK_HEIGHT * Y_DIM * X_DIM * sizeof(bool));
#endif
#endif
    unsigned* trunks_dev;
    hipMalloc(&trunks_dev, BBOX_BLOCK_HEIGHT * Y_DIM * MAX_TRUNK_SIZE * sizeof(unsigned));
    unsigned* trunks_out;
    hipMalloc(&trunks_out, BBOX_BLOCK_HEIGHT * Y_DIM * MAX_TRUNK_SIZE * sizeof(unsigned));
    unsigned* trunk_length;
    hipMalloc(&trunk_length, BBOX_BLOCK_HEIGHT * Y_DIM * sizeof(unsigned));
    hipMemset(trunk_length, 0, BBOX_BLOCK_HEIGHT * Y_DIM * sizeof(unsigned));

#ifdef TEST
    unsigned* trunks_host = (unsigned*)malloc(NUM_LAYERS * MAX_TRUNK_SIZE * Y_DIM * sizeof(unsigned));
#else
    unsigned* trunks_host = (unsigned*)malloc(BBOX_BLOCK_HEIGHT * MAX_TRUNK_SIZE * Y_DIM * sizeof(unsigned));
#endif
    hipMalloc(&triangles_dev, num_triangles * sizeof(triangle));
    hipMemcpy(triangles_dev, triangles.data(), num_triangles * sizeof(triangle), hipMemcpyHostToDevice);
    
    hipMalloc(&points_dev, num_triangles * sizeof(triangle));
    size_t temp_offset = 0;
    for (int i = 0; i < 9; i++) {
        hipMemcpy(points_dev + temp_offset, point_array[i].data(),
                    num_triangles * sizeof(double), hipMemcpyHostToDevice);
        temp_offset += num_triangles;
    }
    hipError_t err = hipGetLastError();  // add
    if (err != hipSuccess) {
        std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl;
        return 1;
    }

    timer_checkpoint(start);
    std::cout << "Slicing...                            ";
    for (unsigned layer_idx = 0; layer_idx < NUM_LAYERS; layer_idx += BBOX_BLOCK_HEIGHT) {
        rectTriIntersection<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>
            (points_dev, num_triangles, trunks_dev, trunk_length, layer_idx);
        hipDeviceSynchronize();
        checkCudaError();
        unsigned max_length = thrust::reduce(thrust::device,
            trunk_length, trunk_length+Y_DIM*BBOX_BLOCK_HEIGHT, 0, thrust::maximum<unsigned>());
        max_length += 2; // Max number of runs + zero terminate
        if (max_length > MAX_TRUNK_SIZE) {
            std::cout << "too many intersections" << std::endl; return 0;
        }
        size_t blocksPerGrid = (Y_DIM * BBOX_BLOCK_HEIGHT + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        trunk_compress<<<blocksPerGrid, THREADS_PER_BLOCK>>>(trunks_dev, trunk_length, trunks_out, max_length);
        hipDeviceSynchronize();
        checkCudaError();
        size_t copy_layers = (layer_idx + BBOX_BLOCK_HEIGHT) < NUM_LAYERS ? BBOX_BLOCK_HEIGHT : NUM_LAYERS - layer_idx;
        size_t copy_size = copy_layers * Y_DIM * max_length * sizeof(unsigned);
        unsigned* trunks_addr = &trunks_host[0];
        hipMemcpy(trunks_addr, trunks_out, copy_size, hipMemcpyDeviceToHost);
        hipMemset(trunk_length, 0, BBOX_BLOCK_HEIGHT * Y_DIM * sizeof(unsigned));
        hipDeviceSynchronize();
        checkCudaError();
    #if (COMPRESSION_ONLY == 0)
    #ifdef TEST
        bool* out_addr = &all[layer_idx*X_DIM*Y_DIM];
    #else
        bool* out_addr = &all[0];
    #endif
        decompression_time += rleDecode(trunks_addr, out_addr, copy_layers, max_length);
    #endif
    }

    timer_checkpoint(start);
    hipFree(trunk_length);
    hipFree(points_dev);
    hipFree(trunks_dev);
    hipFree(trunks_out);

#if (COMPRESSION_ONLY == 0)
    std::cout << "Total decompression time: " << decompression_time << "ms" << std::endl;
#ifdef TEST
    checkOutput(triangles_dev, num_triangles, all);

    // std::ofstream outfile;
    // std::cout << "Writing to output file...                 ";
    // outfile.open("out.txt");
    // for (int z = 0; z < NUM_LAYERS; z++) {
    //     for (int y = Y_DIM-1; y >= 0; y--) {
    //         for (int x = 0; x < X_DIM; x++) {
    //             if (all[z*X_DIM*Y_DIM + y*X_DIM + x]) outfile << "XX";
    //             else outfile << "  ";
    //         }
    //         outfile << "\n";
    //     }
    //     outfile << "\n\n";
    // }
    // outfile.close();
    free(all);
#endif
#endif

    hipFree(triangles_dev);
    free(trunks_host);

    return 0;
}
