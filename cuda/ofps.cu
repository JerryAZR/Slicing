#include <iostream>
#include <fstream>
#include <string>
#include "triangle.cuh"
#include "slicer.cuh"
#include "golden.cuh"
#include <vector>
#include <chrono>
#define NOW (std::chrono::high_resolution_clock::now())

typedef std::chrono::time_point<std::chrono::high_resolution_clock> chrono_t;

void timer_checkpoint(chrono_t & checkpoint) {
#ifdef TEST
    chrono_t end = NOW;
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - checkpoint);
    std::cout << duration.count() << "ms" << std::endl;
    checkpoint = end;
#else
    std::cout << std::endl;
#endif
}

int main(int argc, char* argv[]) {
    std::string stl_file_name;
    std::vector<triangle> triangles;

    if (argc == 2) {
        stl_file_name = argv[1];
    } else if (argc > 2) {
        std::cout << "ERROR: Too many command line arguments" << std::endl;
    }

    chrono_t start = NOW;

    read_stl(stl_file_name,triangles);
    int num_triangles = triangles.size();

    std::cout << "Reading STL file...                   ";
    timer_checkpoint(start);
    std::cout << "Allocating device memory...           ";

    triangle* triangles_dev;
    // all[z][y][x]
    bool* all = (bool*)malloc(NUM_LAYERS * Y_DIM * X_DIM * sizeof(bool));
    bool* all_dev;
    size_t size = NUM_LAYERS * Y_DIM * X_DIM * sizeof(bool);
    hipMalloc(&all_dev, size);
    hipMalloc(&triangles_dev, num_triangles * sizeof(triangle));
    hipMemcpy(triangles_dev, triangles.data(), num_triangles * sizeof(triangle), hipMemcpyHostToDevice);

    hipError_t err = hipGetLastError();  // add
    if (err != hipSuccess) std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl;

    long threadsPerBlock = THREADS_PER_BLOCK;
    long blocksPerGrid;

    layer_t* all_intersections;
    hipMalloc(&all_intersections, Y_DIM * X_DIM * MAX_TRUNK_SIZE * sizeof(layer_t));
    unsigned* trunk_length;
    hipMalloc(&trunk_length, Y_DIM * X_DIM * sizeof(unsigned));
    hipMemset(trunk_length, 0, Y_DIM * X_DIM * sizeof(unsigned));
    int* locks;
    hipMalloc(&locks, Y_DIM * X_DIM * sizeof(int));
    hipMemset(locks, 0, Y_DIM * X_DIM * sizeof(int));

    timer_checkpoint(start);
    std::cout << "Stage 1: Ray Triangle Intersection    ";

    blocksPerGrid = (num_triangles * Y_DIM * X_DIM + threadsPerBlock - 1) / threadsPerBlock;
    fps1<<<blocksPerGrid, threadsPerBlock>>>(&triangles_dev[0], num_triangles, all_intersections, trunk_length, locks);
    hipDeviceSynchronize();

    timer_checkpoint(start);
    std::cout << "Stage 2: Trunk Sorting                ";

    blocksPerGrid = (X_DIM * Y_DIM + threadsPerBlock - 1) / threadsPerBlock;
    fps2<<<blocksPerGrid, threadsPerBlock>>>(all_intersections, trunk_length);
    hipDeviceSynchronize();

    timer_checkpoint(start);
    std::cout << "Stage 3: Layer Extraction             ";

    blocksPerGrid = (X_DIM * Y_DIM * NUM_LAYERS + threadsPerBlock - 1) / threadsPerBlock;
    fps3<<<blocksPerGrid, threadsPerBlock>>>(all_intersections, trunk_length, all_dev);
    hipDeviceSynchronize();

    timer_checkpoint(start);
    std::cout << "Copying memory contents...            ";

    hipFree(all_intersections);
    hipFree(trunk_length);
    hipFree(locks);
    // Copy result from device memory to host memory
    hipMemcpy(all, all_dev, size, hipMemcpyDeviceToHost);
    timer_checkpoint(start);

    err = hipGetLastError();  // add
    if (err != hipSuccess) std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl;
    std::cout << "begin verification" << std::endl;
#ifdef TEST
    checkOutput(triangles_dev, num_triangles, all);
    // std::ofstream outfile;
    // outfile.open("layers.txt");
    // for (int z = 0; z < NUM_LAYERS; z++) {
    //     for (int y = Y_DIM-1; y >= 0; y--) {
    //         for (int x = 0; x < X_DIM; x++) {
    //             if (all[z*Y_DIM*X_DIM + y*X_DIM + x]) outfile << "XX";
    //             else outfile << "  ";
    //         }
    //         outfile << "\n";
    //     }
    //     outfile << "\n\n";
    // }
    // outfile.close();
#endif
    hipFree(all_dev);
    hipFree(triangles_dev);
    free(all);

    return 0;
}
