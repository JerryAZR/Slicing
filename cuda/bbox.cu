#include <iostream>
#include <string>
#include "triangle.cuh"
#include "slicer.cuh"
#include "golden.cuh"
#include <vector>
#include <chrono>
#define NOW (std::chrono::high_resolution_clock::now())

typedef std::chrono::time_point<std::chrono::high_resolution_clock> chrono_t;

void timer_checkpoint(chrono_t & checkpoint) {
#ifdef TEST
    chrono_t end = NOW;
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - checkpoint);
    std::cout << duration.count() << "ms" << std::endl;
    checkpoint = end;
#else
    std::cout << std::endl;
#endif
}
 

int main(int argc, char* argv[]) {
    std::string stl_file_name;
    std::vector<triangle> triangles;
    std::vector<std::vector<double>> point_array(9);

    if (argc == 2) {
        stl_file_name = argv[1];
    } else if (argc > 2) {
        std::cout << "ERROR: Too many command line arguments" << std::endl;
    }

    chrono_t start = NOW;

    load_point_array(stl_file_name, point_array, triangles);

    std::cout << "Reading STL file...                   ";
    timer_checkpoint(start);
    std::cout << "Allocating device memory...           ";

    int num_triangles = triangles.size();
    triangle* triangles_dev;
    double* points_dev;
    // all[z][y][x]
    bool* all = (bool*)malloc(NUM_LAYERS * Y_DIM * X_DIM * sizeof(bool));
    bool* all_dev;
    size_t size = NUM_LAYERS * Y_DIM * X_DIM * sizeof(bool);
    hipMalloc(&all_dev, size);
    hipMemset(all_dev, 0, size);
    hipMalloc(&triangles_dev, num_triangles * sizeof(triangle));
    hipMemcpy(triangles_dev, triangles.data(), num_triangles * sizeof(triangle), hipMemcpyHostToDevice);
    
    hipMalloc(&points_dev, num_triangles * sizeof(triangle));
    size_t temp_offset = 0;
    for (int i = 0; i < 9; i++) {
        hipMemcpy(points_dev + temp_offset, point_array[i].data(),
                    num_triangles * sizeof(double), hipMemcpyHostToDevice);
        temp_offset += num_triangles;
    }
    hipError_t err = hipGetLastError();  // add
    if (err != hipSuccess) {
        std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl;
        return 1;
    }

    timer_checkpoint(start);
    std::cout << "Running 1st kernel...                 ";
    rectTriIntersection<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(points_dev, num_triangles, all_dev);
    hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl;
        return 1;
    }
    timer_checkpoint(start);
    std::cout << "Running 2nd kernel...                 ";
    size_t blocksPerGrid = (X_DIM * Y_DIM + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    layerExtraction<<<blocksPerGrid, THREADS_PER_BLOCK>>>(all_dev, 0);
    hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl;
        return 1;
    }
    timer_checkpoint(start);
    std::cout << "Copying memory contents...            ";
    // Copy result from device memory to host memory
    hipMemcpy(all, all_dev, size, hipMemcpyDeviceToHost);
    err = hipGetLastError();  // add
    if (err != hipSuccess) {
        std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl;
        return 1;
    }
    timer_checkpoint(start);
#ifdef TEST
    checkOutput(triangles_dev, num_triangles, all);
    // for (int z = 0; z < NUM_LAYERS; z++) {
    //     for (int y = Y_DIM; y > 0; y--) {
    //         for (int x = 0; x < X_DIM; x++) {
    //             if (all[z][y][x]) std::cout << "XX";
    //             else std::cout << "  ";
    //         }
    //         std::cout << std::endl;
    //     }
    //     std::cout << std::endl << std::endl;
    // }
#endif
    hipFree(all_dev);
    hipFree(triangles_dev);
    hipFree(points_dev);
    free(all);

    return 0;
}
