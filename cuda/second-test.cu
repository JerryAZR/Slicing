#include <thrust/sort.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <algorithm>
#include <iostream>
#include <string>
#include <vector>
#include <chrono>
#include <utility>
#include "triangle.cuh"
#include "slicer.cuh"
#include "golden.cuh"

#define NOW (std::chrono::system_clock::now())

typedef std::chrono::time_point<std::chrono::system_clock> chrono_t;

void timer_checkpoint(chrono_t & checkpoint) {
    chrono_t end = NOW;
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - checkpoint);
    std::cout << duration.count() << "ms" << std::endl;
    checkpoint = end;
}

int main(int argc, char* argv[]) {
    std::string stl_file_name;
    std::vector<triangle> small_tri;
    std::vector<triangle> large_tri;
    std::vector<double> z_mins_vect;
    chrono_t start;

    if (argc == 2) {
        stl_file_name = argv[1];
    } else if (argc > 2) {
        std::cout << "ERROR: Too many command line arguments" << std::endl;
        return 0;
    } else {
        std::cout << "ERROR: Too few command line arguments" << std::endl;
        return 0;
    }

    start = NOW;
    
    preprocess_stl(stl_file_name, small_tri, large_tri, z_mins_vect);
    size_t num_small = small_tri.size();
    size_t num_large = large_tri.size();

    std::cout << "Reading STL file...           ";
    timer_checkpoint(start);
    std::cout << "Allocating device memory...   ";

    triangle* small_tri_dev;
    hipMalloc(&small_tri_dev, num_small * sizeof(triangle));
    hipMemcpy(small_tri_dev, small_tri.data(), num_small * sizeof(triangle), hipMemcpyHostToDevice);

    triangle* large_tri_dev;
    hipMalloc(&large_tri_dev, num_large * sizeof(triangle));
    hipMemcpy(large_tri_dev, large_tri.data(), num_large * sizeof(triangle), hipMemcpyHostToDevice);

    layer_t* intersections_large;
    hipMalloc(&intersections_large, Y_DIM * X_DIM * NUM_LAYERS * sizeof(layer_t));

    size_t* trunk_length;
    hipMalloc(&trunk_length, Y_DIM * X_DIM * sizeof(size_t));

    // out[y][x][z]
    bool* out = (bool*)malloc(NUM_LAYERS * Y_DIM * X_DIM * sizeof(bool));
    bool* out_dev;
    hipMalloc(&out_dev, Y_DIM * X_DIM * NUM_LAYERS * sizeof(bool));
    hipMemset(out_dev, 0, Y_DIM * X_DIM * NUM_LAYERS * sizeof(bool));

    double* z_mins_dev;
    hipMalloc(&z_mins_dev, num_small * sizeof(double));
    hipMemcpy(z_mins_dev, z_mins_vect.data(), num_small * sizeof(double), hipMemcpyHostToDevice);

    timer_checkpoint(start);
    std::cout << "Sorting triangles...          ";

    // start = std::chrono::system_clock::now();

    // mid = std::chrono::system_clock::now();

    int threadsPerBlock = THREADS_PER_BLOCK;
    int numBlocks;

    numBlocks = (Y_DIM * X_DIM + threadsPerBlock - 1) / threadsPerBlock;

    largeTriIntersection<<<numBlocks, threadsPerBlock>>>(large_tri_dev, num_large, intersections_large, trunk_length);
    thrust::sort_by_key(thrust::device, z_mins_dev, z_mins_dev + num_small, small_tri_dev);

    hipDeviceSynchronize();
    timer_checkpoint(start);
    std::cout << "Processing sorted triangles...";

    smallTriIntersection<<<numBlocks, threadsPerBlock>>>(small_tri_dev, z_mins_dev, num_small, intersections_large, trunk_length, out_dev);

    hipDeviceSynchronize();
    timer_checkpoint(start);
    std::cout << "Copying memory contents...    ";

    hipMemcpy(out, out_dev, Y_DIM * X_DIM * NUM_LAYERS * sizeof(bool), hipMemcpyDeviceToHost);

    timer_checkpoint(start);

    // for (int z = 0; z < NUM_LAYERS; z++) {
    //     for (int y = Y_DIM-1; y >= 0; y--) {
    //         for (int x = 0; x < X_DIM; x++) {
    //             if (out[z][x][y]) std::cout << "XX";
    //             else std::cout << "  ";
    //         }
    //         std::cout << std::endl;
    //     }
    //     std::cout << std::endl << std::endl;
    // }

    small_tri.insert(small_tri.end(), large_tri.begin(), large_tri.end());
    size_t num_triangles = small_tri.size();
    triangle* triangles_dev;
    hipMalloc(&triangles_dev, num_triangles * sizeof(triangle));
    hipMemcpy(triangles_dev, small_tri.data(), num_triangles * sizeof(triangle), hipMemcpyHostToDevice);
    // checkOutput(triangles_dev, num_triangles, out_reshaped);
    checkOutput(triangles_dev, num_triangles, out);

    free(out);
    hipFree(large_tri_dev);
    hipFree(small_tri_dev);
    hipFree(intersections_large);
    hipFree(trunk_length);

    return 0;
}