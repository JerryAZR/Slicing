#include <iostream>
#include <string>
#include "triangle.cuh"
#include "slicer.cuh"
#include "golden.cuh"
#include <vector>


#define PPS 0
#define SHOW_LAYER 0

int main(int argc, char* argv[]) {
    std::string stl_file_name;
    std::vector<triangle> triangles;

    if (argc == 2) {
        stl_file_name = argv[1];
    } else if (argc > 2) {
        std::cout << "ERROR: Too many command line arguments" << std::endl;
    }

    read_stl(stl_file_name,triangles);
    int num_triangles = triangles.size();
    triangle* triangles_dev;
    // all[z][y][x]
    bool all[NUM_LAYERS][Y_DIM][X_DIM];
    bool* all_dev;
    size_t size = NUM_LAYERS * Y_DIM * X_DIM * sizeof(bool);
    hipMalloc(&all_dev, size);
    hipMalloc(&triangles_dev, num_triangles * sizeof(triangle));
    //hipMemcpy(all_dev, &all[0][0][0], size, hipMemcpyHostToDevice); // unnecessary
    hipMemcpy(triangles_dev, triangles.data(), num_triangles * sizeof(triangle), hipMemcpyHostToDevice);

    hipError_t err = hipGetLastError();  // add
    if (err != hipSuccess) std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl;

    int threadsPerBlock = THREADS_PER_BLOCK;
    int blocksPerGrid;

#if(PPS == 1)
    blocksPerGrid = (Y_DIM * X_DIM + threadsPerBlock - 1) / threadsPerBlock;
    pps<<<blocksPerGrid, threadsPerBlock>>>(&triangles_dev[0], num_triangles, all_dev);
    hipDeviceSynchronize();
    err = hipGetLastError();  // add
    if (err != hipSuccess) std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl;
#else
    char* all_intersections;
    hipMalloc(&all_intersections, Y_DIM * X_DIM * NUM_LAYERS * sizeof(char));
    size_t* trunk_length;
    hipMalloc(&trunk_length, Y_DIM * X_DIM * sizeof(size_t));
    hipMemset(trunk_length, 0, Y_DIM * X_DIM * sizeof(size_t));
    int* locks;
    hipMalloc(&locks, Y_DIM * X_DIM * sizeof(int));
    hipMemset(locks, 0, Y_DIM * X_DIM * sizeof(int));

    blocksPerGrid = (num_triangles * Y_DIM * X_DIM + threadsPerBlock - 1) / threadsPerBlock;
    blocksPerGrid = (blocksPerGrid + threadsPerBlock - 1) / threadsPerBlock; // multi triangles per thread;
    fps1<<<blocksPerGrid, threadsPerBlock>>>(&triangles_dev[0], num_triangles, all_intersections, trunk_length, locks);
    hipDeviceSynchronize();
    blocksPerGrid = (X_DIM * Y_DIM + threadsPerBlock - 1) / threadsPerBlock;
    fps2<<<blocksPerGrid, threadsPerBlock>>>(all_intersections, trunk_length);
    hipDeviceSynchronize();
    blocksPerGrid = (X_DIM * Y_DIM * NUM_LAYERS + threadsPerBlock - 1) / threadsPerBlock;
    fps3<<<blocksPerGrid, threadsPerBlock>>>(all_intersections, trunk_length, all_dev);
    hipDeviceSynchronize();

    hipFree(all_intersections);
    hipFree(trunk_length);
    hipFree(locks);
#endif
    // Copy result from device memory to host memory
    hipMemcpy(&all[0][0][0], all_dev, size, hipMemcpyDeviceToHost);
    err = hipGetLastError();  // add
    if (err != hipSuccess) std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl;

    bool expected[NUM_LAYERS][Y_DIM][X_DIM];
    goldenModel(triangles_dev, num_triangles, &expected[0][0][0]);
    long diff = compare(&all[0][0][0], &expected[0][0][0], NUM_LAYERS*Y_DIM*X_DIM);
    std::cout << "diff: " << diff << std::endl;

    hipFree(all_dev);
    hipFree(triangles_dev);

#if (SHOW_LAYER==0)
    return 0; // Skip the following code
#endif
    // Visualize
    for (int y = Y_DIM; y > 0; y--) {
        for (int x = 0; x < X_DIM; x++) {
            if (all[10][y][x]) std::cout << "x";
            else std::cout << " ";
        }
        std::cout << std::endl;
    }

    return 0;
}
