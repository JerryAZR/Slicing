#include "hip/hip_runtime.h"
#include "slicer.cuh"
#include "triangle.cuh"
#include <thrust/functional.h>
#include <thread>

__device__ __forceinline__ void triangleCopy(void* src, void* dest, int id);
__device__ __forceinline__ double min3(double a, double b, double c);
__device__ __forceinline__ double max3(double a, double b, double c);
__device__ __forceinline__ char atomicAdd(char* address, char val);
__device__ __forceinline__ int pixelRayIntersection_point(double x1, double y1, double z1,
    double x2, double y2, double z2, double x3, double y3, double z3, int x, int y);

__global__ void rectTriIntersection(double* tri_global, size_t num_tri, unsigned* trunks, unsigned* trunk_length, unsigned base_layer) {
    size_t idx = (size_t)blockDim.x * (size_t)blockIdx.x + (size_t)threadIdx.x;
    size_t num_per_thread = num_tri / (NUM_BLOCKS << LOG_THREADS) + 1;
    size_t base_idx = idx;

    double* x1_base = tri_global;
    double* y1_base = tri_global + num_tri;
    double* z1_base = tri_global + 2*num_tri;
    double* x2_base = tri_global + 3*num_tri;
    double* y2_base = tri_global + 4*num_tri;
    double* z2_base = tri_global + 5*num_tri;
    double* x3_base = tri_global + 6*num_tri;
    double* y3_base = tri_global + 7*num_tri;
    double* z3_base = tri_global + 8*num_tri;

    // iterate over all triangles assigned to this thread.
    for (size_t i = 0; i < num_per_thread; i++) {
        // Compute bounding box
        if (base_idx >= num_tri) break;
        double x1 = x1_base[base_idx];
        double y1 = y1_base[base_idx];
        double z1 = z1_base[base_idx];
        double x2 = x2_base[base_idx];
        double y2 = y2_base[base_idx];
        double z2 = z2_base[base_idx];
        double x3 = x3_base[base_idx];
        double y3 = y3_base[base_idx];
        double z3 = z3_base[base_idx];
        
        long xMin = __double2ll_ru(min3(x1, x2, x3) / RESOLUTION);
        long zMin = __double2ll_ru(min3(z1, z2, z3) / RESOLUTION);
        long xMax = __double2ll_rd(max3(x1, x2, x3) / RESOLUTION);
        long zMax = __double2ll_rd(max3(z1, z2, z3) / RESOLUTION);
        base_idx += (NUM_BLOCKS << LOG_THREADS);
        // Make sure the bounds are inside the supported space
        xMax = min(xMax, X_MAX);
        xMin = max(xMin, X_MIN);
        long zMax_ub = min(NUM_LAYERS-1, (long)(base_layer+BLOCK_HEIGHT-1));
        zMax = min(zMax, zMax_ub);
        zMin = max(zMin, (long)(base_layer));
        if (xMax < xMin || zMax < zMin) continue;
        // iterate over all pixels inside the bounding box
        // Will likely cause (lots of) wrap divergence, but we'll deal with that later
        int x = xMin;
        int z = zMin;
        while (z <= zMax) {
            int curr_intersection = 
                pixelRayIntersection_point(x1, y1, z1, x2, y2, z2, x3, y3, z3, x, z);
            if (curr_intersection >= Y_MIN && curr_intersection <= Y_MAX) {
                // Found a valid intersection
                int x_idx = x + (X_DIM >> 1);
                unsigned y_idx = curr_intersection + (Y_DIM >> 1);
                // Add current intersection to trunk
                unsigned* trunk_base = trunks + (z-base_layer)*X_DIM*MAX_TRUNK_SIZE + x_idx*MAX_TRUNK_SIZE;
                unsigned* length_address = trunk_length + (z-base_layer)*X_DIM + x_idx;
                unsigned curr_length = atomicAdd(length_address, 1);
                // Need to check if out of range
                if (curr_length >= MAX_TRUNK_SIZE) 
                    printf("Error: Too many intersections.\n \
                            Please increase MAX_TRUNK_SIZE in slicer.cuh and recompile.\n");
                trunk_base[curr_length] = y_idx;
            }
            // update coords
            bool nextLine = (x == xMax);
            z += (int)nextLine;
            x = nextLine ? xMin : (x+1);
        }
    }
}

__global__ void trunk_compress(unsigned* trunks, unsigned* trunk_length) {
    size_t idx = (size_t)blockDim.x * (size_t)blockIdx.x + (size_t)threadIdx.x;
    unsigned length = trunk_length[idx];
    unsigned* trunk_base = trunks + idx*MAX_TRUNK_SIZE;
    bool curr = false;
    bool prev = false;
    unsigned out_length = 0;

    thrust::sort(thrust::device, trunk_base, trunk_base + length);
    trunk_base[length] = Y_DIM;

    unsigned layerIdx = 0;
    for (unsigned y = 0; y < Y_DIM; y++) {
        // update prev flag
        prev = curr;
        // If intersect
        while (trunk_base[layerIdx] < y) layerIdx++;
        bool intersect = (y == trunk_base[layerIdx]);
        bool flag = (bool) (layerIdx & 1);
        curr = intersect || flag;
        if (curr != prev) {
            trunk_base[out_length] = y;
            out_length++;
        }
    }
    if (out_length < MAX_TRUNK_SIZE) trunk_base[out_length] = Y_DIM;
}

// single thread ver
void bbox_ints_decompress_st(unsigned* in, bool* out, unsigned nlayers) {
    for (unsigned z = 0; z < nlayers; z++) {
        for (unsigned x = 0; x < X_DIM; x++) {
            unsigned* in_base = in + (z*X_DIM*MAX_TRUNK_SIZE + x*MAX_TRUNK_SIZE);
            bool* out_base = out + (z*Y_DIM*X_DIM + x);
            unsigned flip_idx = 0;
            bool inside = false;
            for (unsigned y = 0; y < Y_DIM; y++) {
                if (in_base[flip_idx] == y) {
                    inside = !inside;
                    flip_idx++;
                }
                out_base[y*X_DIM] = inside;
            }
        }
    }
}

void bbox_ints_decompress(unsigned* in, bool* out) {
    unsigned num_per_thread = (NUM_LAYERS + NUM_CPU_THREADS - 1) / NUM_CPU_THREADS;
    std::thread threads[NUM_CPU_THREADS];
    size_t in_offset = 0;
    size_t out_offset = 0;
    for (unsigned i = 0; i < NUM_CPU_THREADS-1; i++) {
        unsigned* thread_in = in + in_offset;
        bool* thread_out = out + out_offset;
        threads[i] = std::thread(bbox_ints_decompress_st, thread_in, thread_out, num_per_thread);
        in_offset += (num_per_thread*X_DIM*MAX_TRUNK_SIZE);
        out_offset += (num_per_thread*X_DIM*Y_DIM);
    }
    unsigned remaining = NUM_LAYERS - ((NUM_CPU_THREADS-1)*num_per_thread);
    unsigned* thread_in = in + in_offset;
    bool* thread_out = out + out_offset;
    threads[NUM_CPU_THREADS-1] = std::thread(bbox_ints_decompress_st, thread_in, thread_out, remaining);
    for (unsigned i = 0; i < NUM_CPU_THREADS; i++) {
        threads[i].join();
    }
}


/**
 * pixelRayIntersection: helper function, computes the intersection of given triangle and pixel ray
 * Inputs:
 *      t -- input triangle
 *      x, y -- coordinates of the input pixel ray
 * Returns:
 *      The layer on which they intersect, or -1 if no intersection
 */
__device__ __forceinline__
int pixelRayIntersection_point(double x1, double y1, double z1,
    double x2, double y2, double z2, double x3, double y3, double z3, int x, int z) {
    /*
    Let A, B, C be the 3 vertices of the given triangle
    Let S(x,y,z) be the intersection, where x,y are given
    We want to find some a, b such that AS = a*AB + b*AC
    If a >= 0, b >= 0, and a+b <= 1, S is a valid intersection.
    */

    double x_pos = x * RESOLUTION;
    double z_pos = z * RESOLUTION;

    // double x_max = max3(x1, x2, x3);
    // double x_min = min3(x1, x2, x3);

    // if (x_pos < x_min || x_pos > x_max) return NONE;

    double x_d = x_pos - x1;
    double z_d = z_pos - z1;

    double xx1 = x2 - x1;
    double yy1 = y2 - y1;
    double zz1 = z2 - z1;

    double xx2 = x3 - x1;
    double yy2 = y3 - y1;
    double zz2 = z3 - z1;
    double a = (x_d * zz2 - xx2 * z_d) / (xx1 * zz2 - xx2 * zz1);
    double b = (x_d * zz1 - xx1 * z_d) / (xx2 * zz1 - xx1 * zz2);
    bool inside = (a >= 0) && (b >= 0) && (a+b <= 1);
    double intersection = (a * yy1 + b * yy2) + y1;
    // // divide by layer width
    int layer = inside ? (intersection / RESOLUTION) : INT_MIN;
    return layer;
}
 
// Copy (THREADS_PER_BLOCK) triangles from src to dest
// Achieves 100% memory efficiency
__device__ __forceinline__
void triangleCopy(void* src, void* dest, int id) {
    copy_unit_t* src_ptr = (copy_unit_t*) src;
    copy_unit_t* dest_ptr = (copy_unit_t*) dest;

    #pragma unroll
    for (int d = 0; d < unit_per_tri; d++) {
        size_t offset = d * THREADS_PER_BLOCK;
        dest_ptr[id + offset] = src_ptr[id + offset];
    }
}

__device__ __forceinline__
double min3(double a, double b, double c) {
    // thrust::minimum<double> min;
    return min(a, min(b, c));
}

__device__ __forceinline__
double max3(double a, double b, double c) {
    // thrust::maximum<double> max;
    return max(a, max(b, c));
}
