#include "hip/hip_runtime.h"
#include "slicer.cuh"
#include "triangle.cuh"
#include <thrust/functional.h>
#include <thread>
#include <chrono>
#define NOW (std::chrono::high_resolution_clock::now())
typedef std::chrono::time_point<std::chrono::high_resolution_clock> chrono_t;

__device__ __forceinline__ void triangleCopy(void* src, void* dest, int id);
__device__ __forceinline__ double min3(double a, double b, double c);
__device__ __forceinline__ double max3(double a, double b, double c);
__device__ __forceinline__ char atomicAdd(char* address, char val);
__device__ __forceinline__ int pixelRayIntersection_point(double x1, double y1, double z1,
    double x2, double y2, double z2, double x3, double y3, double z3, int x, int y);

__global__ void rectTriIntersection(double* tri_global, size_t num_tri, unsigned* trunks, unsigned* trunk_length, unsigned base_layer) {
    size_t idx = (size_t)blockDim.x * (size_t)blockIdx.x + (size_t)threadIdx.x;
    size_t num_per_thread = num_tri / (NUM_BLOCKS << LOG_THREADS) + 1;
    size_t base_idx = idx;

    double* x1_base = tri_global;
    double* y1_base = tri_global + num_tri;
    double* z1_base = tri_global + 2*num_tri;
    double* x2_base = tri_global + 3*num_tri;
    double* y2_base = tri_global + 4*num_tri;
    double* z2_base = tri_global + 5*num_tri;
    double* x3_base = tri_global + 6*num_tri;
    double* y3_base = tri_global + 7*num_tri;
    double* z3_base = tri_global + 8*num_tri;

    // iterate over all triangles assigned to this thread.
    for (size_t i = 0; i < num_per_thread; i++) {
        // Compute bounding box
        if (base_idx >= num_tri) break;
        double x1 = x1_base[base_idx];
        double y1 = y1_base[base_idx];
        double z1 = z1_base[base_idx];
        double x2 = x2_base[base_idx];
        double y2 = y2_base[base_idx];
        double z2 = z2_base[base_idx];
        double x3 = x3_base[base_idx];
        double y3 = y3_base[base_idx];
        double z3 = z3_base[base_idx];
        
        long yMin = __double2ll_ru(min3(y1, y2, y3) / RESOLUTION);
        long zMin = __double2ll_ru(min3(z1, z2, z3) / RESOLUTION);
        long yMax = __double2ll_rd(max3(y1, y2, y3) / RESOLUTION);
        long zMax = __double2ll_rd(max3(z1, z2, z3) / RESOLUTION);
        base_idx += (NUM_BLOCKS << LOG_THREADS);
        // Make sure the bounds are inside the supported space
        yMax = min(yMax, Y_MAX);
        yMin = max(yMin, Y_MIN);
        long zMax_ub = min(NUM_LAYERS-1, (long)(base_layer+BLOCK_HEIGHT-1));
        zMax = min(zMax, zMax_ub);
        zMin = max(zMin, (long)(base_layer));
        if (yMax < yMin || zMax < zMin) continue;
        // iterate over all pixels inside the bounding box
        // Will likely cause (lots of) wrap divergence, but we'll deal with that later
        int y = yMin;
        int z = zMin;
        while (z <= zMax) {
            int curr_intersection = 
                pixelRayIntersection_point(x1, y1, z1, x2, y2, z2, x3, y3, z3, y, z);
            if (curr_intersection >= X_MIN && curr_intersection <= X_MAX) {
                // Found a valid intersection
                int y_idx = y + (Y_DIM >> 1);
                unsigned x_idx = curr_intersection + (X_DIM >> 1);
                // Add current intersection to trunk
                unsigned* trunk_base = trunks + (z-base_layer)*Y_DIM*MAX_TRUNK_SIZE + y_idx;
                unsigned* length_address = trunk_length + (z-base_layer)*Y_DIM + y_idx;
                unsigned curr_length = atomicAdd(length_address, 1);
                // Need to check if out of range
                if (curr_length >= MAX_TRUNK_SIZE) 
                    printf("Error: Too many intersections.\n \
                            Please increase MAX_TRUNK_SIZE in slicer.cuh and recompile.\n");
                trunk_base[curr_length*Y_DIM] = x_idx;
            }
            // update coords
            bool nextLine = (y == yMax);
            z += (int)nextLine;
            y = nextLine ? yMin : (y+1);
        }
    }
}

__global__ void trunk_compress(unsigned* trunks, unsigned* trunk_length, unsigned* out) {
    size_t idx = (size_t)blockDim.x * (size_t)blockIdx.x + (size_t)threadIdx.x;
    size_t y_idx = idx % Y_DIM;
    size_t z_idx = idx / Y_DIM;
    unsigned length = trunk_length[idx];
    unsigned* trunk_base = out + idx*MAX_TRUNK_SIZE;
    unsigned out_length = 0;
    unsigned prev_idx = 0;

    unsigned input_trunk[MAX_TRUNK_SIZE];
    for (unsigned i = 0; i < length; i++) {
        input_trunk[i] = *(trunks + z_idx*MAX_TRUNK_SIZE*Y_DIM + i*Y_DIM + y_idx);
    }
    __syncthreads();
    thrust::sort(thrust::device, input_trunk, input_trunk + length);
    if (length < MAX_TRUNK_SIZE) input_trunk[length] = X_DIM;

    unsigned i = 0;
    // Manually process the first intersection to avoid problems
    trunk_base[out_length++] = input_trunk[0];
    prev_idx = input_trunk[0];
    i = 0;

    while (i < length) {
        // Find the next run of 1's
        i++;
        while ((input_trunk[i] - input_trunk[i-1] <= 1 || i & 1 == 1) && i < length) {
            i++;
        }
        unsigned run_1s = input_trunk[i-1] - prev_idx + 1;
        unsigned run_0s = (i == length) ?
                X_DIM - input_trunk[i-1] - 1 : input_trunk[i] - input_trunk[i-1] - 1;
        prev_idx = input_trunk[i];
        trunk_base[out_length++] = run_1s;
        trunk_base[out_length++] = run_0s;
    }
    if (out_length < MAX_TRUNK_SIZE) trunk_base[out_length] = 0;
}

// single thread ver
void bbox_ints_decompress_st(unsigned* in, bool* out, unsigned nlayers) {
    for (unsigned z = 0; z < nlayers; z++) {
        for (unsigned y = 0; y < Y_DIM; y++) {
            // if (z == 1) printf("STarting layer %d, row %d.\n", z, y);
            unsigned* in_base = in + (z*Y_DIM*MAX_TRUNK_SIZE + y*MAX_TRUNK_SIZE);
            bool* out_base = out + (z*Y_DIM*X_DIM + y*X_DIM);
            bool inside = false;
            unsigned start = 0;
            unsigned length;
            for (unsigned idx = 0; in_base[idx] != 0; idx++) {
                length = in_base[idx];
                memset(out_base+start, inside, length);
                inside = !inside;
                start += length;
            }
        }
    }
}

// Returns the running time
double bbox_ints_decompress(unsigned* in, bool* out, unsigned nlayers) {
    chrono_t start = NOW;
    
    unsigned num_per_thread = (nlayers + NUM_CPU_THREADS - 1) / NUM_CPU_THREADS;
    std::thread threads[NUM_CPU_THREADS];
    size_t in_offset = 0;
    size_t out_offset = 0;
    for (unsigned i = 0; i < NUM_CPU_THREADS-1; i++) {
        unsigned* thread_in = in + in_offset;
        bool* thread_out = out + out_offset;
        threads[i] = std::thread(bbox_ints_decompress_st, thread_in, thread_out, num_per_thread);
        in_offset += (num_per_thread*X_DIM*MAX_TRUNK_SIZE);
        out_offset += (num_per_thread*X_DIM*Y_DIM);
    }
    unsigned remaining = nlayers - ((NUM_CPU_THREADS-1)*num_per_thread);
    unsigned* thread_in = in + in_offset;
    bool* thread_out = out + out_offset;
    threads[NUM_CPU_THREADS-1] = std::thread(bbox_ints_decompress_st, thread_in, thread_out, remaining);
    for (unsigned i = 0; i < NUM_CPU_THREADS; i++) {
        threads[i].join();
    }

    auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(NOW - start);
    double ms = (double)duration.count() / 1e6;
    return ms;
}


/**
 * pixelRayIntersection: helper function, computes the intersection of given triangle and pixel ray
 * Inputs:
 *      t -- input triangle
 *      x, y -- coordinates of the input pixel ray
 * Returns:
 *      The layer on which they intersect, or -1 if no intersection
 */
__device__ __forceinline__
int pixelRayIntersection_point(double x1, double y1, double z1,
    double x2, double y2, double z2, double x3, double y3, double z3, int y, int z) {
    /*
    Let A, B, C be the 3 vertices of the given triangle
    Let S(x,y,z) be the intersection, where x,y are given
    We want to find some a, b such that AS = a*AB + b*AC
    If a >= 0, b >= 0, and a+b <= 1, S is a valid intersection.
    */

    double y_pos = y * RESOLUTION;
    double z_pos = z * RESOLUTION;

    double y_d = y_pos - y1;
    double z_d = z_pos - z1;

    double xx1 = x2 - x1;
    double yy1 = y2 - y1;
    double zz1 = z2 - z1;

    double xx2 = x3 - x1;
    double yy2 = y3 - y1;
    double zz2 = z3 - z1;
    double a = (y_d * zz2 - yy2 * z_d) / (yy1 * zz2 - yy2 * zz1);
    double b = (y_d * zz1 - yy1 * z_d) / (yy2 * zz1 - yy1 * zz2);
    bool inside = (a >= 0) && (b >= 0) && (a+b <= 1);
    double intersection = (a * xx1 + b * xx2) + x1;
    // // divide by layer width
    int layer = inside ? (intersection / RESOLUTION) : INT_MIN;
    return layer;
}

__device__ __forceinline__
double min3(double a, double b, double c) {
    // thrust::minimum<double> min;
    return min(a, min(b, c));
}

__device__ __forceinline__
double max3(double a, double b, double c) {
    // thrust::maximum<double> max;
    return max(a, max(b, c));
}
